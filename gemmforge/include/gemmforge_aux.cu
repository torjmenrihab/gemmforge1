#include <iostream>
#include <hip/hip_runtime.h>

namespace gemmforge {
    std::string PrevFile = "";
    int PrevLine = 0;

    void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << File 
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error) 
                  << " (" << Error << ")" 
                  << std::endl;
                  
        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = File;
      PrevLine = Line;
#endif
    }

  void synchDevice(void *stream) {
    hipDeviceSynchronize();
    checkErr(__FILE__, __LINE__);
  }
}


